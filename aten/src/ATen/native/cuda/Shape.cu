#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/Resize.h>
#include <ATen/native/TypeProperties.h>
#include <ATen/native/TensorShape.h>
#include <ATen/Dispatch.h>
#include <c10/core/MemoryFormat.h>
#include <c10/util/Optional.h>

namespace at {
namespace native {

#if defined(USE_ROCM)
constexpr int CAT_ARRAY_BATCH_SIZE = 1024;
#else
constexpr int CAT_ARRAY_BATCH_SIZE = 128;
#endif
constexpr int CAT_ARRAY_MAX_INPUT_DIMS = 4;

namespace {

inline bool getCatGrid(ptrdiff_t nTensors, dim3& grid) {
  const int numSM = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

  //X dim of grid for cat array cooperates on a single tensor in the cat.
  //Given half of the GPU, full utilization will always occur.
  grid = dim3( 2LL * numSM, (long long) nTensors );

  return true;
}

// Similar to any other IndexToOffset calculation for copying along a given
// dimension.
template <typename IndexType, int Dims>
struct CatArrIndexToOffset {
  static inline __device__ IndexType compute(
      const IndexType tensorSize[Dims],
      const IndexType tensorStride[Dims],
      const IndexType dimSize,
      const unsigned int concatDim,
      IndexType linearIndex) {
    // linearIndex is not really linear index, but instead the offset in
    // input tensor. If the input tensor is contiguous, then this offset
    // is the linear index, but if the input tensor is channels last, then
    // it is the linear index of the permuted contiguous tensor
    IndexType offset = 0;

#pragma unroll
    for (int i = Dims - 1; i >= 1; --i) {
      IndexType curDimSize = i == concatDim ? dimSize : tensorSize[i];
      IndexType nextDimIndex = linearIndex / curDimSize;
      IndexType curDimIndex = linearIndex - curDimSize * nextDimIndex;
      IndexType curDimOffset = curDimIndex * tensorStride[i];
      offset += curDimOffset;
      linearIndex = nextDimIndex;
    }

    return offset + linearIndex * tensorStride[0];
  }
};

template<typename IndexType, unsigned int MaxDims>
struct TensorSizeStride {
  IndexType tensorSize[MaxDims];
  IndexType tensorStride[MaxDims];
};

/**
  * Kernel used to concatenated grimDim.y tensors into an output tensor. Uses a
  * grid-stride loop based off of the blockIdx.x, threadIdx.x for each input to
  * copy each element from each input tensor into the output.
  *
  * output: base pointer to the storage associated with the output tensor
  * inputs: GPU-allocated array of input metadata for each input to concatenate
  *         in the kernel
  * os: the size/stride vectors for the output tensor
  * concatDim: dimension along which we are concatenating
  * dimStride: the stride of the output tensor at the concatDim
  *
  * The most important assumption made is that the input tensors are contiguous.
  */


// Use pinned memory and and pass the struct by pointer on ROCm
template <typename T, typename IndexType>
struct CatArrInputTensor {
  T* input;
  IndexType offset;
  IndexType dimSize;
  IndexType nElements;
};

template <typename T, typename IndexType, int Dims>
C10_LAUNCH_BOUNDS_1(512)
__global__ void HIP_CatArrayBatchedCopy(
    T* output,
    CatArrInputTensor<T, IndexType>* inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs[blockIdx.y].nElements;

    if(tid >= nElements) return;

    T* data = inputs[blockIdx.y].input;
    IndexType offset = inputs[blockIdx.y].offset;
    IndexType dimSize = inputs[blockIdx.y].dimSize;
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
    IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                  os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
    output[dataOffset + elementOffset] = data[tid];

    tid += stride;
    }
}

// pass meta data directly through kernel argument instead of pin memory
// In contiguous case, we will not need stride_size, setting it as 1 as placeholder
// to pass compile.
template <typename T, typename IndexType, int n, int stride_size>
struct CatArrInputTensorMetadata {
  T* input[n];
  IndexType offset[n];
  IndexType dimSize[n];
  IndexType nElements[n];
  bool isContiguous[n];
  TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> tensorStride[stride_size];
};

template <typename T, typename IndexType, int Dims, int batch_size, int stride_size>
__global__ void CatArrayBatchedCopy(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> ins = stride_size > 1 ? inputs.tensorStride[blockIdx.y] : inputs.tensorStride[0];
    bool isContig = inputs.isContiguous[blockIdx.y];

    if(tid >= nElements) return;

    T* data = inputs.input[blockIdx.y];
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
      IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
      if (isContig) {
        output[dataOffset + elementOffset] = data[tid];
      } else {
        IndexType inElementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    ins.tensorSize, ins.tensorStride, dimSize, concatDim, tid);
        output[dataOffset + elementOffset] = data[inElementOffset];
      }
    tid += stride;
    }
}

template <typename scalar_t>
void hip_parallel_cat(const Tensor &out, ITensorList inputs, int64_t dimension,
                      int nDims, c10::MemoryFormat memory_format) {
  // First, let's set up our kernel parameters. We start with a raw pointer to
  // the storage for the output Tensor.
  scalar_t *data = out.data_ptr<scalar_t>();

  // Kernel Parameter
  long tensorMetadataSize =
    sizeof(CatArrInputTensor<scalar_t, unsigned int>) * CAT_ARRAY_BATCH_SIZE;
  auto d_inputs_storage = at::empty(
    {tensorMetadataSize}, out.options().dtype(at::kByte));
  auto d_inputs = static_cast<CatArrInputTensor<scalar_t, unsigned int> *>(
    d_inputs_storage.data_ptr());

  TensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> outputParam;

  // Next, let's initialize the size, stride arrays for the output Tensor.
  if (memory_format == c10::MemoryFormat::Contiguous) {
    for (int i = 0; i < nDims; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i);
      outputParam.tensorStride[i] = out.stride(i);
    }
  } else if (memory_format == c10::MemoryFormat::ChannelsLast || memory_format == c10::MemoryFormat::ChannelsLast3d) {
    // permute the semantics of dims from NCHW to NHWC so that the input
    // tensor is now contiguous
    outputParam.tensorSize[0] = at::native::size(out, 0);
    outputParam.tensorStride[0] = out.stride(0);
    for (int i = 1; i < nDims - 1; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i + 1);
      outputParam.tensorStride[i] = out.stride(i + 1);
    }
    outputParam.tensorSize[nDims - 1] = at::native::size(out, 1);
    outputParam.tensorStride[nDims - 1] = out.stride(1);
  } else {
    TORCH_CHECK(false, "unsupported memory format");
  }

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  // Now we loop
  int batchCounter = 0;
  int64_t offset = 0;
  for (int i = 0; i < inputs.size() ; i += CAT_ARRAY_BATCH_SIZE) {
    // Re-allocate stackInputs every iteration to avoid read-after-write hazard
    {
      auto stackInputs_storage = at::empty({tensorMetadataSize},
          out.options().dtype(at::kByte).device(at::kCPU).pinned_memory(true));
      auto stackInputs =
        static_cast<CatArrInputTensor<scalar_t, unsigned int> *>(
          stackInputs_storage.data_ptr());
      for (batchCounter = 0;
           batchCounter < CAT_ARRAY_BATCH_SIZE &&
             (i+batchCounter) < inputs.size();
           ++batchCounter) {
        int64_t dimSize = 0;
        // There is a legacy case where a 1-D empty tensor can be concat with
        // high-dimensional tensor
        if (inputs[i+batchCounter].numel() > 0) {
          dimSize = at::native::size(inputs[i+batchCounter], dimension);
        }

        stackInputs[batchCounter].input =
          inputs[i+batchCounter].data_ptr<scalar_t>();
        stackInputs[batchCounter].offset = offset;
        stackInputs[batchCounter].dimSize = dimSize;
        stackInputs[batchCounter].nElements = inputs[i+batchCounter].numel();

        // update offset
        offset += dimSize;
      }
      at::native::copy_(d_inputs_storage, stackInputs_storage,
                        /* non_blocking= */ true);
    }

    // Next, let's consider how we set our kernel launch parameters.
    // We borrow from THCApply, which the kernel's internal indexing
    // is based on.
    dim3 applyBlock = dim3(32*16);

    //Get grid where x dim fills half gpu and y dim is number of tensors.
    //This will have cating two tensors fill the entire grid, but prevent
    //many threads from needlessly load meta data if their sizes is small.
    dim3 catGrid;
    getCatGrid(batchCounter, catGrid);

    if (memory_format != c10::MemoryFormat::Contiguous) {
      switch (dimension) {
      case 0:
        break;
      case 1:
        dimension = nDims - dimension;
        break;
      default:
        dimension--;
      }
    }
    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
    HIP_CatArrayBatchedCopy<scalar_t, unsigned int, DIMS><<<\
        catGrid, applyBlock, 0, stream.stream()>>>(\
            data, d_inputs, outputParam, dimension, outputParam.tensorStride[dimension]); \
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    switch (nDims) {
      case 1:
        HANDLE_CASE(1);
        break;
      case 2:
        HANDLE_CASE(2);
        break;
      case 3:
        HANDLE_CASE(3);
        break;
      case 4:
        HANDLE_CASE(4);
        break;
    }
#undef HANDLE_CASE
  }
}

template <typename scalar_t, int batch_size, int stride_size>
void parallel_cat(const Tensor &out, ITensorList inputs, int64_t dimension,
                  int nDims, c10::MemoryFormat memory_format) {
  // First, let's set up our kernel parameters. We start with a raw pointer to
  // the storage for the output Tensor.
  scalar_t *data = out.data_ptr<scalar_t>();
  CatArrInputTensorMetadata<scalar_t, unsigned int, batch_size, stride_size> catMetaData;
  TensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> outputParam;

  // Next, let's initialize the size, stride arrays for the output Tensor.
  if (memory_format == c10::MemoryFormat::Contiguous) {
    for (int i = 0; i < nDims; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i);
      outputParam.tensorStride[i] = out.stride(i);
    }
  } else if (memory_format == c10::MemoryFormat::ChannelsLast || memory_format == c10::MemoryFormat::ChannelsLast3d) {
    // permute the semantics of dims from NCHW to NHWC so that the input
    // tensor is now contiguous
    outputParam.tensorSize[0] = at::native::size(out, 0);
    outputParam.tensorStride[0] = out.stride(0);
    for (int i = 1; i < nDims - 1; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i + 1);
      outputParam.tensorStride[i] = out.stride(i + 1);
    }
    outputParam.tensorSize[nDims - 1] = at::native::size(out, 1);
    outputParam.tensorStride[nDims - 1] = out.stride(1);
  } else {
    TORCH_CHECK(false, "unsupported memory format");
  }

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  // Now we loop
  int batchCounter = 0;
  int64_t offset = 0;
  for (int i = 0; i < inputs.size() ; i += batch_size) {
    for (batchCounter = 0;
          batchCounter < batch_size &&
            (i+batchCounter) < inputs.size();
          ++batchCounter) {
      int64_t dimSize = 0;
      // There is a legacy case where a 1-D empty tensor can be concat with
      // high-dimensional tensor
      if (inputs[i+batchCounter].numel() > 0) {
        dimSize = at::native::size(inputs[i+batchCounter], dimension);
      }
      catMetaData.input[batchCounter] = inputs[i+batchCounter].data_ptr<scalar_t>();
      catMetaData.offset[batchCounter] = offset;
      catMetaData.dimSize[batchCounter] = dimSize;
      catMetaData.nElements[batchCounter] = inputs[i+batchCounter].numel();
      if (stride_size > 1) {
        auto strides = inputs[i+batchCounter].strides();
        auto sizes = inputs[i+batchCounter].sizes();
        for(int j = 0; j < nDims; j++){
          catMetaData.tensorStride[batchCounter].tensorSize[j] = sizes[j];
          catMetaData.tensorStride[batchCounter].tensorStride[j] = strides[j];
        }
        catMetaData.isContiguous[batchCounter] = false;
      } else {
        catMetaData.isContiguous[batchCounter] = true;
      }
      // update offset
      offset += dimSize;
    }
    // Next, let's consider how we set our kernel launch parameters.
    // We borrow from THCApply, which the kernel's internal indexing
    // is based on.
    dim3 applyBlock = dim3(32*16);

    //Get grid where x dim fills half gpu and y dim is number of tensors.
    //This will have cating two tensors fill the entire grid, but prevent
    //many threads from needlessly load meta data if their sizes is small.
    dim3 catGrid;
    getCatGrid(batchCounter, catGrid);

    if (memory_format != c10::MemoryFormat::Contiguous) {
      switch (dimension) {
      case 0:
        break;
      case 1:
        dimension = nDims - dimension;
        break;
      default:
        dimension--;
      }
    }
    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
    CatArrayBatchedCopy<scalar_t, unsigned int, DIMS, batch_size, stride_size><<<\
        catGrid, applyBlock, 0, stream.stream()>>>(\
            data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]); \
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    switch (nDims) {
      case 1:
        HANDLE_CASE(1);
        break;
      case 2:
        HANDLE_CASE(2);
        break;
      case 3:
        HANDLE_CASE(3);
        break;
      case 4:
        HANDLE_CASE(4);
        break;
    }
#undef HANDLE_CASE
  }
}
} // namespace

TORCH_IMPL_FUNC(cat_out_cuda)
(const ITensorList& tensors,
 int64_t dim,
 int64_t valid,
 bool all_contiguous,
 bool all_same_dtype,
 bool all_same_sizes_and_stride,
 MemoryFormat memory_format,
 const Tensor& result) {
  if (result.numel() == 0) {
    return;
  }

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. The out tensor is 32-bit indexable
  // 3. The number of dimensions is <= 4
  // 4. All input tensors are contiguous (output tensor may be non-contig)
  // 5. All input tensors can use 32-bit indexing

  const bool all32BitIndexable = std::all_of(tensors.begin(), tensors.end(),
    [] (const Tensor& t) {
      return at::cuda::detail::canUse32BitIndexMath(t);
    });

  int nDims = tensors[valid].dim();

#if defined(USE_ROCM)
  if (tensors.size() > 1 &&
      result.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(result) &&
      all_contiguous &&
      all32BitIndexable &&
      all_same_dtype) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          result.scalar_type(), "cat_cuda", [&]() {
        hip_parallel_cat<scalar_t>(result, tensors, dim, nDims, memory_format);
      });
#else
  // We support the contiguous inputs and non-contiguous input (<=4 dims) in different ways
  // For contiguous input, we don't need to pass stride meta data to cuda kernel through constant
  // memory. Therefore, we could pass more inputs to cuda threads.
  // For non-contiguous, we reduce the number of inputs passed to cuda kernel due to the limitation
  // of constant memory.
  if (tensors.size() > 1 &&
      result.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(result) &&
      all_contiguous &&
      all32BitIndexable &&
      all_same_dtype) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          result.scalar_type(), "cat_cuda", [&]() {
        parallel_cat<scalar_t, CAT_ARRAY_BATCH_SIZE, 1>(result, tensors, dim, nDims, memory_format);
      });
  } else if (tensors.size() > 1 &&
      result.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(result) &&
      nDims <= CAT_ARRAY_MAX_INPUT_DIMS &&
      all32BitIndexable &&
      all_same_dtype &&
      memory_format == c10::MemoryFormat::Contiguous) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          result.scalar_type(), "cat_cuda", [&]() {
        parallel_cat<scalar_t, CAT_ARRAY_BATCH_SIZE/2, CAT_ARRAY_BATCH_SIZE/2>(result, tensors, dim, nDims, memory_format);
      });
#endif
  } else {
    int64_t offset = 0;
    for (int j = 0; j < tensors.size(); j++) {
      if (cat_should_skip_tensor(tensors[j])) continue;
      int64_t dimSize = at::native::size(tensors[j], dim);
      Tensor nt = at::narrow(result, dim, offset, dimSize);
      copy_(nt, tensors[j]);
      offset += dimSize;
    }
  }
}

} // namespace native
} // namespace at
